#include "hip/hip_runtime.h"
#include "device.h"
#include "prod_force.h"

template <typename FPTYPE, int THREADS_PER_BLOCK>
__global__ void force_deriv_wrt_center_atom(FPTYPE* force,
                                            const FPTYPE* net_deriv,
                                            const FPTYPE* in_deriv,
                                            const int ndescrpt,
                                            const int nloc,
                                            const int nall) {
  __shared__ FPTYPE data[THREADS_PER_BLOCK * 3];
  int_64 bid = blockIdx.x;
  unsigned int tid = threadIdx.x;
  for (int ii = tid; ii < THREADS_PER_BLOCK * 3; ii += THREADS_PER_BLOCK) {
    data[ii] = (FPTYPE)0.;
  }
  for (int ii = tid; ii < ndescrpt; ii += THREADS_PER_BLOCK) {
    for (int jj = 0; jj < 3; jj++) {
      data[jj * THREADS_PER_BLOCK + tid] +=
          net_deriv[bid * ndescrpt + ii] *
          in_deriv[bid * ndescrpt * 3 + ii * 3 + jj];
    }
  }
  __syncthreads();
  // do reduction in shared memory
  for (int ii = THREADS_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
    if (tid < ii) {
      for (int jj = 0; jj < 3; jj++) {
        data[jj * THREADS_PER_BLOCK + tid] +=
            data[jj * THREADS_PER_BLOCK + tid + ii];
      }
    }
    __syncthreads();
  }
  // write result for this block to global memory
  const int_64 kk = bid / nloc;  // frame index
  const int_64 ll = bid % nloc;  // atom index
  const int_64 i_idx_nall = kk * nall + ll;
  if (tid == 0) {
    force[i_idx_nall * 3 + 0] -= data[THREADS_PER_BLOCK * 0];
    force[i_idx_nall * 3 + 1] -= data[THREADS_PER_BLOCK * 1];
    force[i_idx_nall * 3 + 2] -= data[THREADS_PER_BLOCK * 2];
  }
}

/* compute $\partial E_j / \partial D_{ji} \nabla_R_j D_{ji}$ */
template <typename FPTYPE, bool radial_only_ = true>
__inline__ __device__ void calculate_forces(
    FPTYPE* forces__,
    const FPTYPE* __restrict__ net_deriv__,
    const FPTYPE* __restrict__ in_deriv__,
    const int idx__) {
  const int offset_j = idx__ * ((radial_only_) ? (1) : (4));
  for (int idw = 0; idw < ((radial_only_) ? (1) : (4)); ++idw) {
    const FPTYPE cst1 = net_deriv__[offset_j + idw];
    forces__[0] += cst1 * in_deriv__[(offset_j + idw) * 3 + 0];
    forces__[1] += cst1 * in_deriv__[(offset_j + idw) * 3 + 1];
    forces__[2] += cst1 * in_deriv__[(offset_j + idw) * 3 + 2];
  }
}

/*
  The original code computes

  f_j += \sum_{k neighbor of j} \delta_{ik} f_jk

  Parallelization is over the i index not the j index which means that several
  threads can update the force of the same atom j. It requires an atomic
  operation to do so.

  Instead each thread should update a unique atom $j$ (basically the $f_j$) and
  computes the corresponding term on the right end side of

  f_j += \sum_{k neighbor of j} f_kj.

  In practice, we associate a unique thread block to each atom $j$ while the block
  threads compute the f_kj (note the reverse order in the indices) individually.
  The final sum (over the neighbors k of the atom j) is a block reduction that
  can be done deterministically.
*/

template <typename FPTYPE, bool radial_only_ = true, int shared_memory_block_>
__global__ void force_deriv_wrt_neighbors(
    FPTYPE* force,
    const FPTYPE* net_deriv,
    const FPTYPE* in_deriv,
    const int* nlist,  // neigbhor list
    const int nframes,
    const int nloc,  // number of atoms on this GPU
    const int nall,  // total number of atoms
    const int nnei)  // number fo neighbors
{
  // limited to 2 billions atoms and 2 billions frames
  const int atom_id = blockIdx.x;
  const int frame_id = blockIdx.z * gridDim.y + blockIdx.y;

  if (frame_id >= nframes) {
    return;
  }

  /*
    structure of the arrays:
       - nlist[] -> nlist[frame][atom][neighbor] (in multi dimensional array
    form)
       - net_deriv[] -> net_deriv[frame][atom][neighbor][1 or 4 elements]
       - in_deriv[] -> in_deriv[frame][atom][neighbor][1 or 4 elements][xyz]
       - force[] -> force[frame][all_atoms][xyz]
  */
  const int ndescrpt = nnei * ((radial_only_) ? (1) : (4));

  // define various pointers for a specific frame.
  const FPTYPE* frame_net_deriv_ =
      &net_deriv[frame_id * nloc *
                 ndescrpt];  // f_net_deriv[atom][neighbor][1 or 4]
  const FPTYPE* frame_in_deriv_ =
      &in_deriv[frame_id * nloc * ndescrpt *
                3];  // f_in_deriv[atom][neighbor][1 or 4][3]
  const int* frame_neighbor_list_ =
      &nlist[frame_id * nnei * nloc];  // f_neighbor_list[atom][neighbors]
  FPTYPE force_tmp[3] = {(FPTYPE)0., (FPTYPE)0., (FPTYPE)0.};

  // used to broadcast if a given atom i has some of its neighbors on this GPU.
  // In that case we may need to apply a final reduction at the end of the
  // computation
  //
  // it is always set to 1 when we use one GPU only.
  __shared__ int do_reduction;
  if (threadIdx.x == 0) {
    do_reduction = ((nall == nloc) ? (1) : (0));
  }
  __syncthreads();

  if (nloc != nall) {
    /*
      Each GPU contains a subpart of the entire system.

      - The neighbor list only contains informations about the atoms located on
      that GPU not the entire system.

      - we do not have all needed information to do the computation so we have
      to rely on the same logic than the original implementation using atomic
      operations.

      - To avoid the atomicAdd we treat each atom one by one and follow these
      steps, (i) search if it is a neighbor of one or several local atoms (data
      are on this GPU), (ii) if so calculate the forces and accumulate the
      results locally (iii) when done apply a final reduction and store the
      results back in global memory
     */

    for (int idx = threadIdx.x; idx < nloc * nnei; idx += blockDim.x) {
      if (frame_neighbor_list_[idx] == atom_id) {
        calculate_forces<FPTYPE, radial_only_>(force_tmp, frame_net_deriv_,
                                               frame_in_deriv_, idx);

        // the atomicAdd has NO impact on determinism. It is used to update a
        // shared memory variable value across the thread block to indicate if
        // we need to apply a block reduction at the end of the calculations.
        // It will be different from zero if at least one thread has all
        // informations for the computations. The value should simply be
        // different from zero and updated once. The worst case scenario will
        // be if all threads of a given block have to do the calculations.

        if (do_reduction == 0) {
          atomicAdd(&do_reduction, 1);
        }
        break;
      }
    }
  } else {
    /* Each GPU has the full information about the system. retrieving
       information is easier as we only have to do a linear search over the
       neighbor list of the neighbors not the entire neighbor list. */

    /* each thread selects a neighbor. */
    for (int neighbor_id = threadIdx.x; neighbor_id < nnei;
         neighbor_id += blockDim.x) {
      // collect all terms $\partial E_j / \partial D_{ji} \nabla_R_j D_{ji}$
      // where the atom i is a neighbor of the atom j.
      //
      // Go through all neighbors of atom i, locate the position of
      // the atom i in the neighbor list of the atom j and retrieve all
      // necessary information.

      const int atom_j = frame_neighbor_list_[atom_id * nnei + neighbor_id];

      // The neighbors of a given atom are sorted by type and each resulting
      // list is separated from the other by a series of -1. More details about
      // the sorting can be found in https://doi.org/10.1016/j.cpc.2020.107624
      //
      // To illustrate this, take the neigbhors of a given atom of type a (in a
      // system with two atoms type a and b) deepmd stores the neighbors as
      //
      // [neighbors list of type a], -1, -1, -1, ...., [neighbor list of type
      // b], -1, -1, -1, .....

      if (atom_j < 0) {
        continue;
      }

      const int* nei_nei_list_ = &frame_neighbor_list_[atom_j * nnei];
      int atom_id_position = 0;

      // search the index of the atom i in the local neighbor list of atom j
      for (atom_id_position = 0; atom_id_position < nnei; atom_id_position++) {
        if (nei_nei_list_[atom_id_position] == atom_id) {
          break;
        }
      }
      calculate_forces<FPTYPE, radial_only_>(force_tmp, frame_net_deriv_,
                                             frame_in_deriv_, atom_j * nnei + atom_id_position);
    }
  }

  __syncthreads();

  // Apply the final reduction.
  if (do_reduction) {
    __shared__ FPTYPE fx[shared_memory_block_];
    __shared__ FPTYPE fy[shared_memory_block_];
    __shared__ FPTYPE fz[shared_memory_block_];

    fx[threadIdx.x] = force_tmp[0];
    fy[threadIdx.x] = force_tmp[1];
    fz[threadIdx.x] = force_tmp[2];
    __syncthreads();

    // do the final reduction
    for (int tt = shared_memory_block_ / 2; tt > 0; tt >>= 1) {
      if (threadIdx.x < tt) {
        fx[threadIdx.x] += fx[threadIdx.x + tt];
        fy[threadIdx.x] += fy[threadIdx.x + tt];
        fz[threadIdx.x] += fz[threadIdx.x + tt];
      }
      __syncthreads();
    }

    /* Note the sign difference between the formula in the PRL paper and the
       code. it is due to \nabla_R_j D_{ji} = -\nabla_R_i D_{ji} */
    if (threadIdx.x == 0) {
      const int64_t offset = (frame_id * nall + atom_id) * 3;
      force[offset] += fx[0];
      force[offset + 1] += fy[0];
      force[offset + 2] += fz[0];
    }
  }
}

template <typename FPTYPE, bool radial_only_ = true>
void prod_force_a_r_gpu(FPTYPE* force,
                        const FPTYPE* net_deriv,
                        const FPTYPE* in_deriv,
                        const int* nlist,
                        const int nloc,
                        const int nall,
                        const int nnei,
                        const int nframes) {
  DPErrcheck(gpuGetLastError());
  DPErrcheck(gpuDeviceSynchronize());

  const int ndescrpt = nnei * ((radial_only_) ? (1) : (4));
  DPErrcheck(gpuMemset(force, 0, sizeof(FPTYPE) * nframes * nall * 3));

  force_deriv_wrt_center_atom<FPTYPE, TPB><<<nframes * nloc, TPB>>>(
      force, net_deriv, in_deriv, ndescrpt, nloc, nall);
  DPErrcheck(gpuGetLastError());
  DPErrcheck(gpuDeviceSynchronize());

  dim3 block_grid;

  if (nframes < 0xffff) {
    block_grid = dim3(nall, nframes, 1);
  } else {
    const int sqrt_nframes = sqrt(nframes);
    block_grid = dim3(nall, sqrt_nframes + 1, sqrt_nframes + 1);
  }
  // to accomodate AMD GPU
  dim3 thread_grid(64, 1, 1);
  force_deriv_wrt_neighbors<FPTYPE, radial_only_, 64>
      <<<block_grid, thread_grid>>>(force, net_deriv, in_deriv, nlist, nframes,
                                    nloc, nall, nnei);
  DPErrcheck(gpuGetLastError());
  DPErrcheck(gpuDeviceSynchronize());
}

namespace deepmd {
template <typename FPTYPE>
void prod_force_a_gpu(FPTYPE* force,
                      const FPTYPE* net_deriv,
                      const FPTYPE* in_deriv,
                      const int* nlist,
                      const int nloc,
                      const int nall,
                      const int nnei,
                      const int nframes) {
  prod_force_a_r_gpu<FPTYPE, false>(force, net_deriv, in_deriv, nlist, nloc,
                                    nall, nnei, nframes);
}

template <typename FPTYPE>
void prod_force_r_gpu(FPTYPE* force,
                      const FPTYPE* net_deriv,
                      const FPTYPE* in_deriv,
                      const int* nlist,
                      const int nloc,
                      const int nall,
                      const int nnei,
                      const int nframes) {
  prod_force_a_r_gpu<FPTYPE, true>(force, net_deriv, in_deriv, nlist, nloc,
                                   nall, nnei, nframes);
}

template void prod_force_a_gpu<float>(float* force,
                                      const float* net_deriv,
                                      const float* in_deriv,
                                      const int* nlist,
                                      const int nloc,
                                      const int nall,
                                      const int nnei,
                                      const int nframes);
template void prod_force_a_gpu<double>(double* force,
                                       const double* net_deriv,
                                       const double* in_deriv,
                                       const int* nlist,
                                       const int nloc,
                                       const int nall,
                                       const int nnei,
                                       const int nframes);
template void prod_force_r_gpu<float>(float* force,
                                      const float* net_deriv,
                                      const float* in_deriv,
                                      const int* nlist,
                                      const int nloc,
                                      const int nall,
                                      const int nnei,
                                      const int nframes);
template void prod_force_r_gpu<double>(double* force,
                                       const double* net_deriv,
                                       const double* in_deriv,
                                       const int* nlist,
                                       const int nloc,
                                       const int nall,
                                       const int nnei,
                                       const int nframes);
}  // namespace deepmd
